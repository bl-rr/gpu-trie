#include "hip/hip_runtime.h"
// src/TrieGPU.cu

#include "TrieGPU.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void initTrieKernel(TrieNode *trie)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MAX_NODES)
    {
        for (int i = 0; i < ALPHABET_SIZE; ++i)
        {
            trie[idx].children[i] = -1;
        }
        trie[idx].isEndOfWord = false;
    }
}

__global__ void insertKernel(TrieNode *trie, int *nodeCounter, char *d_words, int *d_wordOffsets, int *d_wordLengths, int numWords)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numWords)
        return;

    int offset = d_wordOffsets[idx];
    int length = d_wordLengths[idx];
    int nodeIdx = 0; // Start at root

    for (int i = 0; i < length; ++i)
    {
        int charIdx = d_words[offset + i] - 'a';
        int *childPtr = &trie[nodeIdx].children[charIdx];

        int childIdx = atomicCAS(childPtr, -1, atomicAdd(nodeCounter, 1));
        if (childIdx == -1)
        {
            nodeIdx = *childPtr;
        }
        else
        {
            nodeIdx = childIdx;
        }
    }
    trie[nodeIdx].isEndOfWord = true;
}

__global__ void searchKernel(TrieNode *trie, char *d_queries, int *d_queryOffsets, int *d_queryLengths, char *d_results, int numQueries)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numQueries)
        return;

    int offset = d_queryOffsets[idx];
    int length = d_queryLengths[idx];
    int nodeIdx = 0;

    for (int i = 0; i < length; ++i)
    {
        int charIdx = d_queries[offset + i] - 'a';
        nodeIdx = trie[nodeIdx].children[charIdx];
        if (nodeIdx == -1)
        {
            d_results[idx] = 0; // Not found
            return;
        }
    }
    d_results[idx] = trie[nodeIdx].isEndOfWord ? 1 : 0;
}

// Constructor
TrieGPU::TrieGPU()
{
    hipMalloc((void **)&d_trie, MAX_NODES * sizeof(TrieNode));
    hipMalloc((void **)&d_nodeCounter, sizeof(int));

    // Initialize d_nodeCounter to 1 on the device
    int one = 1;
    hipMemcpy(d_nodeCounter, &one, sizeof(int), hipMemcpyHostToDevice);

    // Initialize trie nodes
    int threadsPerBlock = 256;
    int blocksPerGrid = (MAX_NODES + threadsPerBlock - 1) / threadsPerBlock;
    initTrieKernel<<<blocksPerGrid, threadsPerBlock>>>(d_trie);
    hipDeviceSynchronize();
}

// Destructor
TrieGPU::~TrieGPU()
{
    hipFree(d_trie);
    hipFree(d_nodeCounter);
}

void TrieGPU::insertWords(const std::vector<std::string> &words)
{
    int numWords = words.size();

    // Flatten words into a single array
    std::string allWords;
    std::vector<int> wordOffsets(numWords);
    std::vector<int> wordLengths(numWords);
    int offset = 0;
    for (size_t i = 0; i < words.size(); ++i)
    {
        allWords += words[i];
        wordOffsets[i] = offset;
        wordLengths[i] = words[i].size();
        offset += words[i].size();
    }

    // Copy data to device
    char *d_words;
    int *d_wordOffsets;
    int *d_wordLengths;
    hipMalloc((void **)&d_words, allWords.size() * sizeof(char));
    hipMalloc((void **)&d_wordOffsets, numWords * sizeof(int));
    hipMalloc((void **)&d_wordLengths, numWords * sizeof(int));

    hipMemcpy(d_words, allWords.c_str(), allWords.size() * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_wordOffsets, wordOffsets.data(), numWords * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wordLengths, wordLengths.data(), numWords * sizeof(int), hipMemcpyHostToDevice);

    // Launch the insert kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numWords + threadsPerBlock - 1) / threadsPerBlock;
    insertKernel<<<blocksPerGrid, threadsPerBlock>>>(d_trie, d_nodeCounter, d_words, d_wordOffsets, d_wordLengths, numWords);
    hipDeviceSynchronize();

    // Free device memory
    hipFree(d_words);
    hipFree(d_wordOffsets);
    hipFree(d_wordLengths);
}

std::vector<char> TrieGPU::searchWords(const std::vector<std::string> &queries)
{
    int numQueries = queries.size();

    // Flatten queries into a single array
    std::string allQueries;
    std::vector<int> queryOffsets(numQueries);
    std::vector<int> queryLengths(numQueries);
    int offset = 0;
    for (size_t i = 0; i < queries.size(); ++i)
    {
        allQueries += queries[i];
        queryOffsets[i] = offset;
        queryLengths[i] = queries[i].size();
        offset += queries[i].size();
    }

    // Copy query data to device
    char *d_queries;
    int *d_queryOffsets;
    int *d_queryLengths;
    char *d_results;
    hipMalloc((void **)&d_queries, allQueries.size() * sizeof(char));
    hipMalloc((void **)&d_queryOffsets, numQueries * sizeof(int));
    hipMalloc((void **)&d_queryLengths, numQueries * sizeof(int));
    hipMalloc((void **)&d_results, numQueries * sizeof(char));

    hipMemcpy(d_queries, allQueries.c_str(), allQueries.size() * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_queryOffsets, queryOffsets.data(), numQueries * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_queryLengths, queryLengths.data(), numQueries * sizeof(int), hipMemcpyHostToDevice);

    // Launch the search kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numQueries + threadsPerBlock - 1) / threadsPerBlock;
    searchKernel<<<blocksPerGrid, threadsPerBlock>>>(d_trie, d_queries, d_queryOffsets, d_queryLengths, d_results, numQueries);
    hipDeviceSynchronize();

    // Retrieve results
    std::vector<char> h_results(numQueries);
    hipMemcpy(h_results.data(), d_results, numQueries * sizeof(char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_queries);
    hipFree(d_queryOffsets);
    hipFree(d_queryLengths);
    hipFree(d_results);

    return h_results;
}